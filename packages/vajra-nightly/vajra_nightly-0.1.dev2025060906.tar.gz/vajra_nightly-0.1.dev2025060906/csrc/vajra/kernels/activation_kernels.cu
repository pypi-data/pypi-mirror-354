#include "hip/hip_runtime.h"
#include <ATen/cuda/HIPContext.h>
#include <torch/all.h>

#include "kernels/dispatch_utils.h"

namespace vajra {

template<typename T>
__device__ __forceinline__ T silu(const T& x) {
  // x * sigmoid(x)
  return (T) (((float) x) / (1.0f + expf((float) -x)));
}

template<typename scalar_t>
__global__ void silu_and_mul_kernel(
  scalar_t* __restrict__ out,               // [num_tokens, d]
  const scalar_t* __restrict__ input,       // [num_tokens, 2, d]
  const int d) {
  const int64_t token_idx = blockIdx.x;
  for (int64_t idx = threadIdx.x; idx < d; idx += blockDim.x) {
    const scalar_t x = __ldg(&input[token_idx * 2 * d + idx]);
    const scalar_t y = __ldg(&input[token_idx * 2 * d + d + idx]);
    out[token_idx * d + idx] = silu(x) * y;
  }
}

} // namespace vajra

void silu_and_mul(
  torch::Tensor& out,      // [num_tokens, d]
  const torch::Tensor& input)    // [num_tokens, 2 * d]
{
  int64_t num_tokens = input.size(0);
  int d = input.size(1) / 2;

  dim3 grid(num_tokens);
  dim3 block(std::min(d, 1024));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  VAJRA_DISPATCH_FLOATING_TYPES(
    input.scalar_type(),
    "silu_and_mul_kernel",
    [&] {
      vajra::silu_and_mul_kernel<scalar_t><<<grid, block, 0, stream>>>(
        out.data_ptr<scalar_t>(),
        input.data_ptr<scalar_t>(),
        d);
    });
}

namespace vajra {

// Element-wise activation kernel template.
template<typename scalar_t, scalar_t (*ACT_FN)(const scalar_t&)>
__global__ void activation_kernel(
  scalar_t* __restrict__ out,               // [num_tokens, d]
  const scalar_t* __restrict__ input,       // [num_tokens, d]
  const int d) {
  const int64_t token_idx = blockIdx.x;
  for (int64_t idx = threadIdx.x; idx < d; idx += blockDim.x) {
    const scalar_t x = __ldg(&input[token_idx * d + idx]);
    out[token_idx * d + idx] = ACT_FN(x);
  }
}

} // namespace vajra

// Launch element-wise activation kernel.
#define LAUNCH_ACTIVATION_KERNEL(KERNEL)                                                  \
  int64_t num_tokens = input.size(0);                                                         \
  int d = input.size(1);                                                                  \
  dim3 grid(num_tokens);                                                                  \
  dim3 block(std::min(d, 1024));                                                          \
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();                           \
  VAJRA_DISPATCH_FLOATING_TYPES(                                                           \
    input.scalar_type(),                                                                  \
    "activation_kernel",                                                                  \
    [&] {                                                                                 \
      vajra::activation_kernel<scalar_t, KERNEL<scalar_t>><<<grid, block, 0, stream>>>(    \
        out.data_ptr<scalar_t>(),                                                         \
        input.data_ptr<scalar_t>(),                                                       \
        d);                                                                               \
    });

namespace vajra {

template<typename T>
__device__ __forceinline__ T gelu_new_kernel(const T& x) {
  const float x3 = (float) (x * x * x);
  const T t = (T) tanhf((T) (0.79788456f * (float) (x + (T) (0.044715f * x3))));
  return ((T) 0.5) * x * (((T) 1.0) + t);
}

template<typename T>
__device__ __forceinline__ T gelu_fast_kernel(const T& x) {
  const float f = (float) x;
  const T t = (T) tanhf(((T) (f * 0.79788456f)) * (((T) 1.0) + (T) (0.044715f * f) * x));
  return ((T) 0.5) * x * (((T) 1.0) + t);
}

} // namespace vajra

void gelu_new(
  torch::Tensor& out,     // [num_tokens, d]
  const torch::Tensor& input)   // [num_tokens, d]
{
  LAUNCH_ACTIVATION_KERNEL(vajra::gelu_new_kernel);
}

void gelu_fast(
  torch::Tensor& out,     // [num_tokens, d]
  const torch::Tensor& input)   // [num_tokens, d]
{
  LAUNCH_ACTIVATION_KERNEL(vajra::gelu_fast_kernel);
}
